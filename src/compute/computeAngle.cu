
#include <hip/hip_runtime.h>
#include <cstdint>
#include <cstdio>
#include <cstring>
#include <cmath>
#include <fstream>
#include <iostream>
#include <stdexcept>
#include <string>
#include <vector>
#include <limits>
#include <algorithm>
#include <numeric>
#include <iomanip>
#include <cctype>
#include <unordered_map>

// ------------------------ Basic data structs ------------------------
struct Atom {
  float x, y, z;
};

struct Box {
  double Lx{0}, Ly{0}, Lz{0}; // orthorhombic lengths
};

struct Frame {
  std::vector<Atom> atoms;
  Box box;
};

struct Angle { 
  float zloc;  // z location of terminal carbon
  float ccCos; // angle from +z axis of terminal cc bond
  float chCos; // angle from +z axis of the mean terminal ch bonds
};

struct Targets {
  int t; // terminal 
  int c; // carbon next to terminal
  int h1; // hydrogen1 at terminal
  int h2;
  int h3;
  Targets operator+(int k) const { return {t+k, c+k, h1+k, h2+k, h3+k}; }
  Targets& operator+=(int k) { t+=k; c+=k; h1+=k; h2+=k; h3+=k; return *this; }
};

// ------------------------ Molecule Target Selection -------------
enum class Key { DES, DBS, DEO, DEA, Unknown };
Key toKey(std::string s) {
  for (char& ch : s ) ch = std::toupper(static_cast<unsigned char>(ch));
  if (s == "DES") return Key::DES;
  if (s == "DBS") return Key::DBS;
  if (s == "DEO") return Key::DEO;
  if (s == "DEA") return Key::DEA;
  return Key::Unknown;
}

Targets findTargets(const std::string& s, bool left) {
  Targets init{};
  switch ( toKey(s)) {
    case Key::DES:
      init = left ? Targets{1, 2, 19, 20, 21} : Targets{18, 17, 44, 43, 42}; break;
    case Key::DBS:
      init = left ? Targets{1, 2, 23, 24, 25} : Targets{22, 21, 56, 55, 54}; break;
    case Key::DEO:
      init = left ? Targets{1, 2, 11, 12, 13} : Targets{10, 9, 20, 19, 18}; break;
    case Key::DEA:
      init = left ? Targets{1, 2, 15, 16, 17} : Targets{14, 13, 32, 31, 30}; break;
    default:
      throw std::runtime_error("Unknown key");
  }

  init += -1;
  return init;
}
// ----------------------- START READ LAMMPS INPUT ---------------------
// One row from the Atoms section (fields you asked for)
struct AtomRow {
  int index = 0;   // LAMMPS atom id (1-based)
  int mol   = 0;   // molecule id (0 if missing)
  int type  = 0;   // atom type
  double charge = 0.0; // charge (0 if missing)
};

// Parsed result container
struct LammpsData {
  std::vector<AtomRow> atoms;                 // all Atoms rows we read
  std::unordered_map<int, double> masses;     // type -> mass (from Masses)
  std::string atomsStyleHint;                 // e.g., "full", "atomic", "charge", if seen
};

// --------------------- Small utilities ---------------------

// Trim leading/trailing whitespace in-place.
static inline void trimInplace(std::string& s) {
  size_t i = 0;
  while (i < s.size() && std::isspace(static_cast<unsigned char>(s[i]))) ++i;
  size_t j = s.size();
  while (j > i && std::isspace(static_cast<unsigned char>(s[j-1]))) --j;
  s.assign(s, i, j - i);
}

// Remove everything after a '#' (LAMMPS inline comment).
static inline void stripComment(std::string& s) {
  size_t pos = s.find('#');
  if (pos != std::string::npos) s.erase(pos);
}

// Case-insensitive prefix check.
static inline bool startsWithCi(const std::string& s, const std::string& prefix) {
  if (s.size() < prefix.size()) return false;
  for (size_t i = 0; i < prefix.size(); ++i) {
    if (std::tolower(static_cast<unsigned char>(s[i])) !=
        std::tolower(static_cast<unsigned char>(prefix[i]))) return false;
  }
  return true;
}

// Strict integer parse (entire token must be an int).
static inline bool parseIntStrict(const std::string& tok, int& out) {
  const char* p = tok.c_str();
  char* end = nullptr;
  long v = std::strtol(p, &end, 10);
  if (end == p || *end != '\0') return false;
  out = static_cast<int>(v);
  return true;
}

// Strict double parse (entire token must be a number).
static inline bool parseDoubleStrict(const std::string& tok, double& out) {
  const char* p = tok.c_str();
  char* end = nullptr;
  double v = std::strtod(p, &end);
  if (end == p || *end != '\0') return false;
  out = v;
  return true;
}

// Tokenize a whitespace string into vector<string>.
static inline std::vector<std::string> splitWs(const std::string& s) {
  std::vector<std::string> out;
  std::istringstream iss(s);
  std::string w;
  while (iss >> w) out.push_back(w);
  return out;
}

// Heuristic: does a cleaned line look like a known section header (besides Atoms/Masses)?
static inline bool isOtherSectionHeader(const std::string& s) {
  static const char* names[] = {
    "Velocities",
    "Bonds", "Angles", "Dihedrals", "Impropers",
    "Pair Coeffs", "Bond Coeffs", "Angle Coeffs",
    "Dihedral Coeffs", "Improper Coeffs", "PairIJ Coeffs",
    "Atoms", "Masses"  // we will handle these explicitly elsewhere
  };
  for (const char* n : names) {
    if (startsWithCi(s, n)) return true;
  }
  // Fallback: alphabetic line that isn't data (be conservative)
  if (!s.empty() && std::isalpha(static_cast<unsigned char>(s[0]))) return true;
  return false;
}

// Try to extract a style hint from a header line like "Atoms # full"
static inline std::string extractStyleHint(const std::string& headerLine) {
  size_t hash = headerLine.find('#');
  if (hash == std::string::npos) return std::string();
  std::string tail = headerLine.substr(hash + 1);
  trimInplace(tail);
  // Tail might be like: "full" or "charge" or "molecular"
  // Strip trailing words like "atoms" if present (rare)
  return tail;
}

// --------------------- Atoms parsing ---------------------

// Parse one Atoms data line to (index, mol, type, charge) using style rules.
// - If style is known ("full", "molecular", "charge", "atomic"), follow that.
// - If style is empty, infer from token pattern; missing fields default to 0 / 0.0.
static inline bool parseAtomsRow(const std::vector<std::string>& tok,
                                 const std::string& styleHint,
                                 AtomRow& out)
{
  // Defensive defaults
  AtomRow row;

  // Known styles first
  if (!styleHint.empty()) {
    // Atoms # full : id mol type q x y z ...
    if (startsWithCi(styleHint, "full")) {
      if (tok.size() < 4) return false;
      if (!parseIntStrict(tok[0], row.index)) return false;
      if (!parseIntStrict(tok[1], row.mol))   return false;
      if (!parseIntStrict(tok[2], row.type))  return false;
      if (!parseDoubleStrict(tok[3], row.charge)) return false;
      out = row; return true;
    }
    // Atoms # molecular : id mol type x y z ...
    if (startsWithCi(styleHint, "molecular")) {
      if (tok.size() < 3) return false;
      if (!parseIntStrict(tok[0], row.index)) return false;
      if (!parseIntStrict(tok[1], row.mol))   return false;
      if (!parseIntStrict(tok[2], row.type))  return false;
      row.charge = 0.0; out = row; return true;
    }
    // Atoms # charge : id type q x y z ...
    if (startsWithCi(styleHint, "charge")) {
      if (tok.size() < 3) return false;
      if (!parseIntStrict(tok[0], row.index)) return false;
      if (!parseIntStrict(tok[1], row.type))  return false;
      if (!parseDoubleStrict(tok[2], row.charge)) return false;
      row.mol = 0; out = row; return true;
    }
    // Atoms # atomic : id type x y z ...
    if (startsWithCi(styleHint, "atomic")) {
      if (tok.size() < 2) return false;
      if (!parseIntStrict(tok[0], row.index)) return false;
      if (!parseIntStrict(tok[1], row.type))  return false;
      row.mol = 0; row.charge = 0.0; out = row; return true;
    }
    // Unknown hint → fall through to inference
  }

  // No hint (or unrecognized): infer:
  // Try "full"-like first: id, mol, type, charge, ...
  bool ok = false;
  if (tok.size() >= 4) {
    AtomRow t{};
    ok = parseIntStrict(tok[0], t.index)
      && parseIntStrict(tok[1], t.mol)
      && parseIntStrict(tok[2], t.type)
      && parseDoubleStrict(tok[3], t.charge);
    if (ok) { out = t; return true; }
  }
  // Try "charge"-like: id, type, charge, ...
  if (tok.size() >= 3) {
    AtomRow t{};
    ok = parseIntStrict(tok[0], t.index)
      && parseIntStrict(tok[1], t.type)
      && parseDoubleStrict(tok[2], t.charge);
    if (ok) { t.mol = 0; out = t; return true; }
  }
  // Try "atomic"-like: id, type, ...
  if (tok.size() >= 2) {
    AtomRow t{};
    ok = parseIntStrict(tok[0], t.index)
      && parseIntStrict(tok[1], t.type);
    if (ok) { t.mol = 0; t.charge = 0.0; out = t; return true; }
  }

  return false; // couldn’t parse
}

// --------------------- File reader ---------------------

// Parse a LAMMPS data file, collecting Atoms (id,mol,type,charge) and Masses (type->mass).
LammpsData readLammpsData(const std::string& path) {
  std::ifstream fin(path);
  if (!fin) throw std::runtime_error("Cannot open file: " + path);

  LammpsData out;
  enum State { None, InAtoms, InMasses };
  State state = None;

  std::string line;
  while (std::getline(fin, line)) {
    // Keep a copy for header detection; but strip comments for data parsing.
    std::string raw = line;
    stripComment(line);
    trimInplace(line);
    if (line.empty()) continue;

    // Section enters
    if (startsWithCi(raw, "Atoms")) {
      state = InAtoms;
      out.atomsStyleHint = extractStyleHint(raw); // may be empty
      continue;
    }
    if (startsWithCi(raw, "Masses")) {
      state = InMasses;
      continue;
    }

    // If a new (other) section begins, leave current section.
    if (isOtherSectionHeader(raw)) {
      state = None;
      continue;
    }

    // Parse content by section
    if (state == InAtoms) {
      // Tokenize current Atoms data row
      std::vector<std::string> tok = splitWs(line);
      if (tok.empty()) continue;

      AtomRow row;
      if (parseAtomsRow(tok, out.atomsStyleHint, row)) {
        out.atoms.push_back(row);
      } else {
        // Silent skip or log a warning:
        // std::cerr << "Warn: could not parse Atoms line: " << raw << "\n";
      }
      continue;
    }

    if (state == InMasses) {
      // Expect: "type mass" (possibly more tokens we ignore)
      std::vector<std::string> tok = splitWs(line);
      if (tok.size() < 2) continue;
      int type = 0; double mass = 0.0;
      if (parseIntStrict(tok[0], type) && parseDoubleStrict(tok[1], mass)) {
        out.masses[type] = mass; // last one wins if duplicated
      }
      continue;
    }

    // Otherwise (state == None): outside sections → ignore
  }

  return out;
}


double massFromType(const std::vector<std::pair<int, double>>& type2mass, int type, double def = -1.0) {
  auto it = std::lower_bound(type2mass.begin(), type2mass.end(), type, [](const auto& p, int key){return p.first < key; });
  return (it != type2mass.end() && it->first == type) ? it->second : def;
}

float wrapPBC(float x, float L) { // wrap x into [0, L)
  float y = std::fmod(x, L);
  if (y<0) y+=L;
  return y;
}

float comZ(const std::vector<Atom>& atoms, const std::vector<int>& atomTypes, const std::vector<std::pair<int, double>>& type2mass){
  if (atoms.size() != atomTypes.size() ) {
    throw std::runtime_error("atoms/types size mismatch");
  }
  long double M = 0.0L, sz = 0.0L;
  for (size_t i=0; i<atoms.size(); i++) {
    const double m = massFromType(type2mass, atomTypes[i]);
    sz += m*atoms[i].z;
    M += m;
  }
  return (M == 0.0L) ? 0.0 : float(sz / M);
}

float comZPBC(const std::vector<Atom>& atoms, const std::vector<int>& atomTypes, const std::vector<std::pair<int, double>>& type2mass, float Lz){
  if (atoms.size() != atomTypes.size() ) {
    throw std::runtime_error("atoms/types size mismatch");
  }
  if(!(Lz > 0.0) ) {
    throw std::runtime_error("Lz must be positive");
  }
  long double C = 0.0L, S = 0.0L, M=0.0L;
  const long double twoPi = 2.0L * acosl(-1.0L);
  const long double k = twoPi / Lz;

  for (size_t i =0; i< atoms.size(); i++) {
    const double m = massFromType(type2mass, atomTypes[i]);
    const long double theta = k* wrapPBC(atoms[i].z, Lz);
    C += m * std::cos(theta);
    S += m * std::sin(theta);
    M += m;
  }
  if (M == 0.0L) return 0.0;

  long double ang = std::atan2(S, C);
  if (ang < 0) ang += twoPi;
  return float (ang * (Lz / twoPi));
}


void writeLammpsDump(std::ostream& out, long timestep, const std::vector<Atom>& atoms, const std::vector<int> atomTypes, const Box& box) {
  out << "ITEM: TIMESTEP\n" << timestep << "\n";
  out << "ITEM: NUMBER OF ATOMS\n" << atoms.size() << "\n";
  out << "ITEM: BOX BOUNDS pp pp pp\n";
  out << std::setprecision(10) << std::fixed;
  float bl=0;
  out << bl << " " << box.Lx << "\n";
  out << bl << " " << box.Ly << "\n";
  out << bl << " " << box.Lz << "\n";

  out << "ITEM: ATOMS id type x y z\n";
  out << std::setprecision(10) << std::fixed;
  for( int atom=0; atom< atoms.size(); atom++ ) {
    out << atom+1 << " " << atomTypes[atom] << " " << atoms[atom].x << " " <<  atoms[atom].y << " " << atoms[atom].z << "\n"; 
  }
}




// ------------------------ START READ DSD FILE -------------------------------
static inline uint32_t bswap32(uint32_t x) {
  return ((x & 0xFF000000u) >> 24) |
         ((x & 0x00FF0000u) >> 8)  |
         ((x & 0x0000FF00u) << 8)  |
         ((x & 0x000000FFu) << 24);
}
static inline uint64_t bswap64(uint64_t x) {
  return ((x & 0xFF00000000000000ull) >> 56) |
         ((x & 0x00FF000000000000ull) >> 40) |
         ((x & 0x0000FF0000000000ull) >> 24) |
         ((x & 0x000000FF00000000ull) >> 8)  |
         ((x & 0x00000000FF000000ull) << 8)  |
         ((x & 0x0000000000FF0000ull) << 24) |
         ((x & 0x000000000000FF00ull) << 40) |
         ((x & 0x00000000000000FFull) << 56);
}

template <typename T>
static inline void byteswap_inplace(T& v) {
  if (sizeof(T) == 4) {
    uint32_t t; std::memcpy(&t, &v, 4); t = bswap32(t); std::memcpy(&v, &t, 4);
  } else if (sizeof(T) == 8) {
    uint64_t t; std::memcpy(&t, &v, 8); t = bswap64(t); std::memcpy(&v, &t, 8);
  } else {
    // unsupported size: no-op
  }
}

template <typename T>
static inline void maybe_swap_buffer(T* ptr, size_t n, bool need_swap) {
  if (!need_swap) return;
  for (size_t i = 0; i < n; ++i) byteswap_inplace(ptr[i]);
}

// Read one Fortran unformatted record: [int32 len] [payload] [int32 len]
static std::vector<char> read_fortran_record(std::ifstream& f, bool& need_swap) {
  uint32_t len1 = 0;
  if (!f.read(reinterpret_cast<char*>(&len1), 4)) throw std::runtime_error("Unexpected EOF (record start).");

  uint32_t len = len1;
  if (len != 84 && len != 164 && len > (1u<<26)) { // heuristic to detect swapped length
    uint32_t s = bswap32(len1);
    if (s < (1u<<24)) { need_swap = !need_swap; len = s; }
  }

  std::vector<char> payload(len);
  if (!f.read(payload.data(), len)) throw std::runtime_error("Unexpected EOF (record payload).");

  uint32_t len2 = 0;
  if (!f.read(reinterpret_cast<char*>(&len2), 4)) throw std::runtime_error("Unexpected EOF (record end).");
  if (need_swap) { len1 = bswap32(len1); len2 = bswap32(len2); }
  if (len1 != len2) throw std::runtime_error("Record length mismatch.");
  return payload;
}

// ------------------------ DCD Reader ------------------------
class DCDReader {
public:
  explicit DCDReader(const std::string& path) : ifs_(path, std::ios::binary) {
    if (!ifs_) throw std::runtime_error("Cannot open file: " + path);
    parse_header();
  }

  int natoms() const { return natoms_; }
  int nframes() const { return nset_; }
  bool has_unitcell() const { return has_unitcell_; }
  bool has_fixed_atoms() const { return has_fixed_atoms_; }

  std::vector<Frame> read_all() {
    std::vector<Frame> out;
    out.reserve(nset_ > 0 ? nset_ : 64);

    int frame_index = 0;
    while (ifs_.peek() != std::char_traits<char>::eof()) {
      Frame fr;
      read_one_frame(fr, frame_index);

      // Fixed-atoms: fill unchanged atoms from previous full coords
      if (has_fixed_atoms_) {
        if (frame_index == 0) {
          prev_full_coords_ = fr.atoms; // seed
        } else {
          for (int i = 0; i < natoms_; ++i) {
            if (!is_free_mask_[i]) fr.atoms[i] = prev_full_coords_[i];
          }
          prev_full_coords_ = fr.atoms; // update seed
        }
      }

      out.push_back(std::move(fr));
      ++frame_index;
    }
    return out;
  }

private:
  std::ifstream ifs_;
  bool need_swap_ = false;

  int natoms_ = 0;
  int nset_   = 0;
  bool has_unitcell_ = false;

  // Fixed-atoms bookkeeping (tentative, verified)
  bool has_fixed_atoms_ = false;
  int nfixed_ = 0;
  int nfreat_ = 0;
  std::vector<int> free_idx_;          // 0-based indices of free atoms
  std::vector<uint8_t> is_free_mask_;  // 0/1 mask per atom
  std::vector<Atom> prev_full_coords_; // last full coordinates for fixed-atom fill

  void parse_header() {
    // ---- Record 1: "CORD" + 20 ints ----
    auto rec1 = read_fortran_record(ifs_, need_swap_);
    if (rec1.size() < 4 + 20*4) throw std::runtime_error("DCD header too short.");

    char magic[5] = {0,0,0,0,0};
    std::memcpy(magic, rec1.data(), 4);
    if (std::strncmp(magic, "CORD", 4) != 0) {
      throw std::runtime_error("Not a DCD file (missing 'CORD').");
    }

    const int nctrl = 20;
    const int32_t* icntrl = reinterpret_cast<const int32_t*>(rec1.data() + 4);
    std::vector<int32_t> ctrl(nctrl);
    for (int i = 0; i < nctrl; ++i) {
      int32_t v = icntrl[i];
      if (need_swap_) byteswap_inplace(v);
      ctrl[i] = v;
    }

    nset_ = ctrl[0];
    const int iflag = ctrl[7];
    int nfixed_tentative = ctrl[9];

    has_unitcell_ = (iflag & 0x04) != 0;

    // ---- Record 2: title block ----
    auto rec2 = read_fortran_record(ifs_, need_swap_);
    if (rec2.size() < 4) throw std::runtime_error("Corrupt title block.");

    // ---- Record 3: number of atoms ----
    auto rec3 = read_fortran_record(ifs_, need_swap_);
    if (rec3.size() < 4) throw std::runtime_error("Corrupt atom count block.");
    int32_t n = 0;
    std::memcpy(&n, rec3.data(), 4);
    if (need_swap_) byteswap_inplace(n);
    if (n <= 0) throw std::runtime_error("Invalid atom count in DCD.");
    natoms_ = n;

    // ---- (Tentative) fixed-atoms detection, with verification ----
    has_fixed_atoms_ = false;
    nfixed_ = 0;
    nfreat_ = 0;
    free_idx_.clear();
    is_free_mask_.clear();

    if (nfixed_tentative > 0 && nfixed_tentative < natoms_) {
      // Save position, try to read IFREAT
      std::streampos pos = ifs_.tellg();
      int nfreat_try = natoms_ - nfixed_tentative;
      bool ok = false;

      try {
        auto rec_free = read_fortran_record(ifs_, need_swap_);
        if (rec_free.size() == static_cast<size_t>(nfreat_try * 4)) {
          free_idx_.resize(nfreat_try);
          const int32_t* raw = reinterpret_cast<const int32_t*>(rec_free.data());
          ok = true;
          for (int i = 0; i < nfreat_try; ++i) {
            int32_t idx = raw[i];
            if (need_swap_) byteswap_inplace(idx);
            if (idx <= 0 || idx > natoms_) { ok = false; break; }
            free_idx_[i] = idx - 1; // 1-based -> 0-based
          }
        }
      } catch (...) {
        ok = false;
      }

      if (ok) {
        has_fixed_atoms_ = true;
        nfixed_ = nfixed_tentative;
        nfreat_ = natoms_ - nfixed_;
        is_free_mask_.assign(natoms_, 0);
        for (int k = 0; k < nfreat_; ++k) is_free_mask_[ free_idx_[k] ] = 1;
      } else {
        // Rewind and treat as non-fixed (header field was misleading)
        ifs_.clear();
        ifs_.seekg(pos);
      }
    }
  }

  void read_unit_cell(Box& box) {
    // Some writers use 6 doubles, others 6 floats; accept either.
    auto rec = read_fortran_record(ifs_, need_swap_);
    if (rec.size() == 6*sizeof(double)) {
      double cell[6];
      std::memcpy(cell, rec.data(), 6*sizeof(double));
      maybe_swap_buffer(cell, 6, need_swap_);
      box.Lx = cell[0]; // A
      box.Ly = cell[2]; // B
      box.Lz = cell[4]; // C
    } else if (rec.size() == 6*sizeof(float)) {
      float cellf[6];
      std::memcpy(cellf, rec.data(), 6*sizeof(float));
      maybe_swap_buffer(cellf, 6, need_swap_);
      box.Lx = cellf[0];
      box.Ly = cellf[2];
      box.Lz = cellf[4];
    } else if (rec.size() == 0) {
      // Empty cell block (shouldn't happen): keep zeros
    } else {
      throw std::runtime_error("Unexpected unit cell record size.");
    }
  }

  void read_xyz_full(std::vector<Atom>& atoms) {
    auto recx = read_fortran_record(ifs_, need_swap_);
    auto recy = read_fortran_record(ifs_, need_swap_);
    auto recz = read_fortran_record(ifs_, need_swap_);
    
    const size_t want_f = static_cast<size_t>(natoms_) * sizeof(float);
    const size_t want_d = static_cast<size_t>(natoms_) * sizeof(double);
    
    if (recx.size() == want_f && recy.size() == want_f && recz.size() == want_f) {
      // float payloads
      std::vector<float> buf(natoms_);
      std::memcpy(buf.data(), recx.data(), recx.size());
      maybe_swap_buffer(buf.data(), natoms_, need_swap_);
      for (int i = 0; i < natoms_; ++i) atoms[i].x = buf[i];
    
      std::memcpy(buf.data(), recy.data(), recy.size());
      maybe_swap_buffer(buf.data(), natoms_, need_swap_);
      for (int i = 0; i < natoms_; ++i) atoms[i].y = buf[i];
    
      std::memcpy(buf.data(), recz.data(), recz.size());
      maybe_swap_buffer(buf.data(), natoms_, need_swap_);
      for (int i = 0; i < natoms_; ++i) atoms[i].z = buf[i];
    
    } else if (recx.size() == want_d && recy.size() == want_d && recz.size() == want_d) {
      // double payloads
      std::vector<double> bufD(natoms_);
      std::memcpy(bufD.data(), recx.data(), recx.size());
      maybe_swap_buffer(bufD.data(), natoms_, need_swap_);
      for (int i = 0; i < natoms_; ++i) atoms[i].x = static_cast<float>(bufD[i]);
    
      std::memcpy(bufD.data(), recy.data(), recy.size());
      maybe_swap_buffer(bufD.data(), natoms_, need_swap_);
      for (int i = 0; i < natoms_; ++i) atoms[i].y = static_cast<float>(bufD[i]);
    
      std::memcpy(bufD.data(), recz.data(), recz.size());
      maybe_swap_buffer(bufD.data(), natoms_, need_swap_);
      for (int i = 0; i < natoms_; ++i) atoms[i].z = static_cast<float>(bufD[i]);
    
    } else {
      throw std::runtime_error("XYZ array size mismatch (full): got {" +
                               std::to_string(recx.size()) + "," +
                               std::to_string(recy.size()) + "," +
                               std::to_string(recz.size()) + "} bytes");
    }
  }
  void read_xyz_free(std::vector<Atom>& atoms) {
    auto recx = read_fortran_record(ifs_, need_swap_);
    auto recy = read_fortran_record(ifs_, need_swap_);
    auto recz = read_fortran_record(ifs_, need_swap_);
    
    const size_t want_f = static_cast<size_t>(nfreat_) * sizeof(float);
    const size_t want_d = static_cast<size_t>(nfreat_) * sizeof(double);
    
    if (recx.size() == want_f && recy.size() == want_f && recz.size() == want_f) {
      std::vector<float> buf(nfreat_);
    
      std::memcpy(buf.data(), recx.data(), recx.size());
      maybe_swap_buffer(buf.data(), nfreat_, need_swap_);
      for (int i = 0; i < nfreat_; ++i) atoms[ free_idx_[i] ].x = buf[i];
    
      std::memcpy(buf.data(), recy.data(), recy.size());
      maybe_swap_buffer(buf.data(), nfreat_, need_swap_);
      for (int i = 0; i < nfreat_; ++i) atoms[ free_idx_[i] ].y = buf[i];
    
      std::memcpy(buf.data(), recz.data(), recz.size());
      maybe_swap_buffer(buf.data(), nfreat_, need_swap_);
      for (int i = 0; i < nfreat_; ++i) atoms[ free_idx_[i] ].z = buf[i];
    
    } else if (recx.size() == want_d && recy.size() == want_d && recz.size() == want_d) {
      std::vector<double> bufD(nfreat_);
    
      std::memcpy(bufD.data(), recx.data(), recx.size());
      maybe_swap_buffer(bufD.data(), nfreat_, need_swap_);
      for (int i = 0; i < nfreat_; ++i) atoms[ free_idx_[i] ].x = static_cast<float>(bufD[i]);
    
      std::memcpy(bufD.data(), recy.data(), recy.size());
      maybe_swap_buffer(bufD.data(), nfreat_, need_swap_);
      for (int i = 0; i < nfreat_; ++i) atoms[ free_idx_[i] ].y = static_cast<float>(bufD[i]);
    
      std::memcpy(bufD.data(), recz.data(), recz.size());
      maybe_swap_buffer(bufD.data(), nfreat_, need_swap_);
      for (int i = 0; i < nfreat_; ++i) atoms[ free_idx_[i] ].z = static_cast<float>(bufD[i]);
    
    } else {
      throw std::runtime_error("XYZ array size mismatch (free): got {" +
                               std::to_string(recx.size()) + "," +
                               std::to_string(recy.size()) + "," +
                               std::to_string(recz.size()) + "} bytes");
    }
  }

  static inline void parse_cell_record(const std::vector<char>& rec, bool need_swap, Box& box) {
    double v[6] = {0,0,0,0,0,0};
    
    if (rec.size() == 6*sizeof(double)) {
      std::memcpy(v, rec.data(), 6*sizeof(double));
      maybe_swap_buffer(v, 6, need_swap);
    } else if (rec.size() == 6*sizeof(float)) {
      float vf[6];
      std::memcpy(vf, rec.data(), 6*sizeof(float));
      maybe_swap_buffer(vf, 6, need_swap);
      for (int i = 0; i < 6; ++i) v[i] = vf[i];
    } else {
      throw std::runtime_error("Unexpected unit cell record size.");
    }
    
    // Default CHARMM/NAMD/LAMMPS convention: (A, gamma, B, beta, C, alpha)
    double Lx = v[0];
    double Ly = v[2];
    double Lz = v[4];
    
    // Heuristics for quirky writers:
    // If C came out 0 or tiny, try the last slot (some paths put C at index 5),
    // or any entry that must be a length (e.g., > 180, which cannot be an angle).
    auto is_angle_like = [](double a) {
      // many writers use 0 or ~90 for angles; treat [0..180] as angle-like
      return (a >= 0.0 && a <= 180.0);
    };
    
    if (Lz <= 1e-9) {
      if (v[5] > 1e-9 && (!is_angle_like(v[5]) || v[5] > 180.0)) {
        Lz = v[5];                       // take the last slot if it looks like a length
      } else {
        // As a last resort, scan all positions for a plausible length not used yet
        // Prefer values > 180 (cannot be angle), else largest positive not equal to ~90.
        int candidates[6] = {0,1,2,3,4,5};
        double best = 0.0;
        for (int k : candidates) {
          if (k == 0 || k == 2 || k == 4) continue; // already assigned
          double x = v[k];
          if (x > best && (!is_angle_like(x) || x > 180.0)) best = x;
        }
        if (best > 0.0) Lz = best;
      }
    }
    
    box.Lx = Lx;
    box.Ly = Ly;
    box.Lz = Lz;
  }


  void read_one_frame(Frame& fr, int frame_index) {
    fr.atoms.resize(natoms_);
    fr.box = Box{}; // default zeros
    
    // Read the first record of the frame. It might be:
    //   (a) unit cell (6 floats or 6 doubles), or
    //   (b) X array
    auto rec0 = read_fortran_record(ifs_, need_swap_);
    
    const size_t wantX_f = static_cast<size_t>(natoms_) * sizeof(float);
    const size_t wantX_d = static_cast<size_t>(natoms_) * sizeof(double);
    
    bool rec0_is_cell = (rec0.size() == 6*sizeof(double)) || (rec0.size() == 6*sizeof(float));
    
    std::vector<char> recx, recy, recz;
    
    if (rec0_is_cell) {
      // We discovered a per-frame unit cell even if the header didn't say so
      parse_cell_record(rec0, need_swap_, fr.box);
    
      // Now read the three coord arrays
      recx = read_fortran_record(ifs_, need_swap_);
      recy = read_fortran_record(ifs_, need_swap_);
      recz = read_fortran_record(ifs_, need_swap_);
    } else {
      // No cell block here; rec0 is actually X
      recx = std::move(rec0);
      recy = read_fortran_record(ifs_, need_swap_);
      recz = read_fortran_record(ifs_, need_swap_);
    }
    
    // Accept either float or double coordinate payloads and convert to float
    if (recx.size() == wantX_f && recy.size() == wantX_f && recz.size() == wantX_f) {
      std::vector<float> buf(natoms_);
    
      std::memcpy(buf.data(), recx.data(), recx.size());
      maybe_swap_buffer(buf.data(), natoms_, need_swap_);
      for (int i = 0; i < natoms_; ++i) fr.atoms[i].x = buf[i];
    
      std::memcpy(buf.data(), recy.data(), recy.size());
      maybe_swap_buffer(buf.data(), natoms_, need_swap_);
      for (int i = 0; i < natoms_; ++i) fr.atoms[i].y = buf[i];
    
      std::memcpy(buf.data(), recz.data(), recz.size());
      maybe_swap_buffer(buf.data(), natoms_, need_swap_);
      for (int i = 0; i < natoms_; ++i) fr.atoms[i].z = buf[i];
    
    } else if (recx.size() == wantX_d && recy.size() == wantX_d && recz.size() == wantX_d) {
      std::vector<double> bufD(natoms_);
    
      std::memcpy(bufD.data(), recx.data(), recx.size());
      maybe_swap_buffer(bufD.data(), natoms_, need_swap_);
      for (int i = 0; i < natoms_; ++i) fr.atoms[i].x = static_cast<float>(bufD[i]);
    
      std::memcpy(bufD.data(), recy.data(), recy.size());
      maybe_swap_buffer(bufD.data(), natoms_, need_swap_);
      for (int i = 0; i < natoms_; ++i) fr.atoms[i].y = static_cast<float>(bufD[i]);
    
      std::memcpy(bufD.data(), recz.data(), recz.size());
      maybe_swap_buffer(bufD.data(), natoms_, need_swap_);
      for (int i = 0; i < natoms_; ++i) fr.atoms[i].z = static_cast<float>(bufD[i]);
    
    } else {
      throw std::runtime_error(
        "XYZ array size mismatch (full): got {" +
        std::to_string(recx.size()) + "," +
        std::to_string(recy.size()) + "," +
        std::to_string(recz.size()) + "} bytes");
    }
  }
};
//----------------------- END READ DCD -------------------------------

//------------------------ START STATISTICS ---------------------------
struct Histogram {
  std::vector<float> edges;
  std::vector<float> counts;
  float binWidth(std::size_t i=0) const {
    return edges.size() > 1? (edges[i+1]-edges[i]) : 0.0;
  }
};

Histogram makeHistogram(const std::vector<float>& data, size_t nbins, float minEdge, float maxEdge){
  Histogram h;
  h.edges.resize(nbins + 1);
  h.counts.assign(nbins, 0.0);

  if (nbins == 0) return h;
  if (maxEdge == minEdge) maxEdge = minEdge + 1;

  const float width = (maxEdge - minEdge)/ static_cast<float>(nbins);
  for( size_t i =0; i<= nbins; i++ ) h.edges[i] = minEdge + i* width;

  for(size_t k=0; k<data.size(); k++) {
    float x = data[k];
    long idx = static_cast<long>(std::floor( (x-minEdge) / width));
    if (idx < 0 ) idx =0;
    if (idx > static_cast<long>(nbins)) idx = static_cast<long>(nbins)-1;
    h.counts[static_cast<size_t>(idx)] += 1.0;
  }
  return h;
}

Histogram makeHistogramAuto(const std::vector<float>& data, size_t nbins) {
  if ( data.empty() || nbins == 0) return {};
  auto mm = std::minmax_element(data.begin(), data.end());
  float mn = *mm.first;
  float mx = *mm.second;
  if ( mx == mn ) {mn -= 0.5; mx += 0.5;}

  return makeHistogram(data, nbins, mn, mx);
}
std::vector<float> normalizePDF(const Histogram& h) {
  float N = std::accumulate(h.counts.begin(), h.counts.end(), 0.0);
  std::vector<float> pdf(h.counts.size(), 0.0);
  if ( N > 0 && h.edges.size() >=2) {
    for (size_t i =0; i< h.counts.size(); i++) {
      float bw = h.edges[i+1] - h.edges[i];
      if (bw > 0 ) pdf[i] = h.counts[i] / (N*bw);
    }
  }
  return pdf;
}

void printHistogram( const Histogram& h, std::ostream& os = std::cout) {
  std::vector<float> pdf = normalizePDF(h);
  os.setf(std::ios::fixed);
  os << std::setprecision(10);
  int nr = h.edges.size();
  int half = nr/2;
  for( size_t i =half; i< nr; i++ ) {
    float x = 0.5 * (h.edges[i] + h.edges[i+1]);
    os << x << "\t" << (pdf[i] + pdf[2*half-i-1])/2.0 << "\n";
  }
}

std::vector<float> histPDFatEdges(const Histogram& h) {
  size_t nb = h.counts.size();
  std::vector<float> y; 
  y.assign(nb+1, 0.0);

  float N = std::accumulate(h.counts.begin(), h.counts.end(), 0.0);
  if ( N<=0.0 || h.edges.size() != nb+1 ) return y;

  for (size_t i =0; i< nb ; i++){
    float bw = h.edges[i+1] - h.edges[i];
    y[i] = (bw > 0.0) ? (h.counts[i] / N / bw) : 0.0;
  }
  y[nb] = y[nb - (nb > 0 ? 1: 0)];
  return y;
}

float linInterp(float x0, float y0, float x1, float y1, float x) {
  float dx = x1-x0;
  if (dx==0.0) return 0.5*(y0+y1);
  float t= (x-x0)/dx;
  return y0 + t* (y1-y0);
}

float integratePDF(const Histogram& h, float xi, float xf) {
  if (h.edges.size() < 2 || h.counts.size() +1 != h.edges.size()) return 0.0;
  if (xi > xf ) std::swap(xi, xf);

  const std::vector<float>& x = h.edges;
  std::vector<float> y = histPDFatEdges(h);

  float a = x.front();
  float b = x.back();
  if ( xf <= a || xi >= b) return 0.0;
  xi = std::max(xi, a);
  xf = std::min(xf, b);

  std::vector<float>::const_iterator itL = std::upper_bound(x.begin(), x.end(), xi);
  std::vector<float>::const_iterator itR = std::upper_bound(x.begin(), x.end(), xf);
  size_t j = (itL == x.begin() ? 0 : static_cast<size_t>((itL - x.begin()) -1));
  size_t k = (itR == x.begin() ? 0 : static_cast<size_t>((itR - x.begin()) -1));
  if( j >= x.size() -1) j = x.size() -2;
  if( k >= x.size() -1) k= x.size() -2;
  float y_xi = linInterp(x[j], y[j], x[j+1], y[j+1], xi);
  float y_xf = linInterp(x[k], y[k], x[k+1], y[k+1], xf);
  if ( j== k) { 
    return 0.5 * (y_xi + y_xf) * (xf - xi);
  }

  float area = 0.0;
  area += 0.5 * (y_xi + y[j+1]) * (x[j+1] - xi);

  for ( size_t m = j+1; m< k; m++ ) {
    area += 0.5 * (y[m]+y[m+1]) * (x[m+1] - x[m]);
  }
  area += 0.5 * (y[k] + y_xf) * (xf - x[k]);
  return area;
}


int binIndexFromEdges(float x, const std::vector<float>& edges){
  if( x< edges.front() || x > edges.back() ) return -1;
  if ( x == edges.back()) return int(edges.size())-2;
  std::vector<float>::const_iterator it = std::upper_bound(edges.begin(), edges.end(), x);
  int idx = int (it - edges.begin()) -1;
  if (idx < 0 || idx >= int(edges.size()) -1 ) return -1;
  return idx;
}

struct momentStats {
  std::vector<float> mean, se;
};

struct Hist2D {
  std::vector<float> zedges, aedges;
  std::vector<float> counts;
  int nz = 0, na = 0;
  float totalSamples = 0.0;

  float& at(int iz, int ia) { return counts[size_t(iz) * size_t(na) + size_t(ia)];}
  float  at(int iz, int ia) const { return counts[size_t(iz) * size_t(na) + size_t(ia)];}

  float zwidth(int iz ) const { return zedges[iz+1] - zedges[iz];}
  float awidth(int ia ) const { return aedges[ia+1] - aedges[ia];}

  float zcenter(int iz ) const { return 0.5*(zedges[iz+1] + zedges[iz]);}
  float acenter(int ia ) const { return 0.5*(aedges[ia+1] + aedges[ia]);}

  void normalizeMass() {
    if ( totalSamples <= 0.0) return;
    for (size_t i=0; i< counts.size(); i++) counts[i] /= totalSamples;
  }

  void normalizeDensity() {
    if (totalSamples <= 0.0) return;
    for (int iz =0 ; iz < nz; iz++) {
      for (int ia=0; ia < na; ia++ ) {
        float area = zwidth(iz) * awidth(ia);
        if (area > 0) {
          at(iz, ia) /= (totalSamples * area);
        }
      }
    }
  }

  void printHist(const std::string& path) const {
    FILE* f = std::fopen(path.c_str(), "w");
    if (!f) throw std::runtime_error("Fail to open file: " + path);
    std::fprintf(f, "z, cos(theta)\n");
    for( int iz=0; iz< nz; iz++ ) {
      for (int ia=0; ia<na; ia++ ) {
        std::fprintf(f, "%.5f\t%.5f\t%.5f\n", zcenter(iz), acenter(ia), at(iz, ia));
      }
    }
    std::fclose(f);
  }

  std::vector<float> aSliceDensity(float zq) const {
    int iz = binIndexFromEdges(zq, zedges);
    if ( iz< 0) throw std::runtime_error("z value out of range");
    std::vector<float> pdf (na, 0.0);
    float rowSum = 0.0;
    for (int ia = 0; ia< na; ia++) rowSum += at(iz,ia);
    if (rowSum > 0.0) {
      for( int ia=0; ia < na ; ia++ ) {
        float da = awidth(ia);
        if (da > 0.0) pdf[ia] = at(iz, ia) / rowSum / da;
      }
    }
    return pdf;
  }

  void printZsliceHist(float zq, std::ostream& os = std::cout,int precision=10 ) const {
    int iz = binIndexFromEdges(zq, zedges);
    if( iz<0) throw std::runtime_error("z value is out of range");
    std::vector<float> v = aSliceDensity(zq);
    os.setf(std::ios::fixed);
    os << std::setprecision(precision);
    os << "# z-bin center = " << zcenter(iz) << " (query z = " << zq << ")\n";
    os << "# angle center, pdf\n";
    for( size_t i =0; i< na; i++ ) {
      os << acenter(i) << "\t" << v[i] << "\n";
    }
  }

  std::vector<float> kthMomentByZ(int k) const {
    std::vector<float> out(size_t(nz), std::numeric_limits<double>::quiet_NaN());
    for( int iz = 0; iz < nz ; iz ++ ) {
      long double num = 0.0L, den = 0.0L;
      for( int ia = 0; ia < na; ia++){
        const double ac = acenter(ia);
        long double w = at(iz, ia) * awidth(ia);
        if ( w <= 0.0L) continue;
        num += std::pow(ac, k) * w;
        den += w;
      }
      if (den > 0.0L) out[size_t(iz)] = float(num/den);
    }
    return out;
  }
  momentStats kthMomentStatsByZ(int k, const std::vector<float>& raw) const {
    if (k<0) throw std::runtime_error("k must be >= 0");
    if (zedges.size() != nz +1 || aedges.size() != na+1 || raw.size() != nz*na ) {
      throw std::runtime_error("size mismatch");
    }
    momentStats out;
    out.mean.assign(nz, std::numeric_limits<float>::quiet_NaN());
    out.se.assign(nz, std::numeric_limits<float>::quiet_NaN());

    for(int iz =0; iz < nz; iz++) {
      long double N = 0.0L, sumY=0.0L, sumY2=0.0L;
      for( int ia=0; ia<na; ia++) {
        const size_t idx = size_t(iz) * size_t(na) + size_t(ia);
        const long double n = raw[idx];
        if ( n <= 0.0L) continue;
        const long double ac = acenter(ia);
        const long double y = std::pow(ac, k);
        sumY += n*y;
        sumY2+= n*y*y;
        N += n;
      }
      if ( N >=1.0L) {
        const long double mean = sumY / N;
        out.mean[size_t(iz)] = float(mean);
        if ( N > 1.0L) {
          const long double varY = (sumY2 - N*mean*mean) / (N-1.0L);
          out.se[size_t(iz)] = (varY > 0.0L ) ? float(std::sqrt(varY/N)) : 0.0;
        } else {
          out.se[size_t(iz)] = std::numeric_limits<float>::infinity();
        }
      }
    }
    return out;
  }

  // print first and third moments
  /*
  void printMoments(std::ostream& os = std::cout, int precision=10 ) const {
    auto m1 = kthMomentByZ(1);
    auto m3 = kthMomentByZ(3);
    os.setf(std::ios::fixed);
    os << std::setprecision(precision);
    os << "# z center, first moment, third moment\n";
    int st = static_cast<int>(nz/2);
    for( size_t i =st; i< nz; i++ ) {
      os << zcenter(i) << "\t" << (m1[i] - m1[2*st-i-1])/2.0 << "\t" << (m3[i] - m3[2*st-i-1])/2.0 << "\n";
    }
  }
  */
  void printMoments(std::ostream& os, momentStats& stat1, momentStats& stat3, int precision=10 ) const {
    os.setf(std::ios::fixed);
    os << std::setprecision(precision);
    os << "# z center, first moment, standard error, third moment, standard error\n";
    int st = static_cast<int>(nz/2);
    for( size_t i =st; i< nz; i++ ) {
      os << zcenter(i) << "\t" 
        << (stat1.mean[i] - stat1.mean[2*st-i-1])/2.0 << "\t" << (stat1.se[i] + stat1.se[2*st-i-1])/2.0 << "\t" 
        << (stat3.mean[i] - stat3.mean[2*st-i-1])/2.0 << "\t" << (stat3.se[i] + stat3.se[2*st-i-1])/2.0 << "\n";
    }
  }
};



std::pair<float, float> minmaxComponent(const std::vector<Angle>& a, bool use_z) {
  float lo = std::numeric_limits<float>::infinity();
  float hi = -std::numeric_limits<float>::infinity();
  for( size_t i =0; i<a.size(); i++ ) {
    float v = use_z ? float(a[i].zloc) : float(a[i].chCos);
    if (v < lo) lo = v;
    if (v > hi) hi = v;
  }
  if (!std::isfinite(lo) || !std::isfinite(hi)) { lo = 0.0; hi = 1.0;}
  if (hi==lo) hi = lo + 1;
  return std::pair<float, float> (lo, hi);
}

Hist2D makeHist2D(const std::vector<Angle>& data, int nz, int na) {
  if (nz <= 0 || na <= 0) throw std::runtime_error("nz/na must be positive\n");
  float zmin, zmax;
  auto zmm = minmaxComponent(data, true);
  zmin = zmm.first; zmax = zmm.second;

  float amin, amax;
  auto amm = minmaxComponent(data, false);
  amin = amm.first; amax = amm.second;


  Hist2D H;
  H.nz = nz; H.na = na;
  H.zedges.resize(size_t(nz)+1);
  H.aedges.resize(size_t(na)+1);
  H.counts.assign(size_t(nz)*size_t(na), 0.0);
  float dz = (zmax-zmin)/float(nz);
  float da = (amax-amin)/float(na);
  for(int i = 0; i<=nz; i++) H.zedges[i] = zmin + dz*float(i);
  for(int j = 0; j<=na; j++) H.aedges[j] = amin + da*float(j);
  return H;
}

float wrapMinImage( float x, float L ) {
  return x - L * std::floor((x+0.5*L)/L);
}

Hist2D makeHist2DSymZ(const std::vector<Angle>& data, float center, float Lz, float zBinWidth, int na) {
  if (!(Lz > 0.0) || !(zBinWidth > 0.0) ||  na <= 0) {
    throw std::runtime_error("nz/na must be positive\n");
  }

  // find max absolute displacement from center
  float zmaxDisp = 0.0;
  for (const auto& a : data) {
    float d = wrapMinImage(wrapPBC(a.zloc, Lz) - center, Lz);
    float ab = std::fabs(d);
    if (ab > zmaxDisp) zmaxDisp = ab;
  }
  zmaxDisp = std::min(zmaxDisp, 0.5f * Lz);

  // choose bin count per side
  int nSide = std::max(1, int(std::ceil(zmaxDisp / zBinWidth)));
  float R = nSide * zBinWidth;
  int nz = nSide*2;

  std::cout << "max z : " << zmaxDisp << " and nz : " << nz << "\n";

  float amin, amax;
  auto amm = minmaxComponent(data, false);
  amin = amm.first; amax = amm.second;

  Hist2D H;
  H.nz = nz; H.na = na;
  H.zedges.resize(size_t(nz)+1);
  H.aedges.resize(size_t(na)+1);
  H.counts.assign(size_t(nz)*size_t(na), 0.0);
  float dz = zBinWidth;
  float da = (amax-amin)/float(na);
  for(int i = 0; i<=nz; i++) H.zedges[i] = center-R + dz*float(i);
  for(int j = 0; j<=na; j++) H.aedges[j] = amin + da*float(j);
  return H;
}

void hist2DAccumulate(Hist2D& H, const std::vector<Angle>& data, float weight = 1.0) {
  for (size_t k=0; k< data.size(); k++ ) {
    float z = float(data[k].zloc) ;
    float a = float(data[k].chCos);
    if (!std::isfinite(z) || !std::isfinite(a)) continue;
    int iz = binIndexFromEdges(z, H.zedges);
    int ia = binIndexFromEdges(a, H.aedges);
    if (iz >= 0 && ia >= 0) {
      H.at(iz, ia) += weight;
      H.totalSamples += weight;
    }
  }
}

void hist2DAdd(Hist2D& H, float z, float a, float weight=1.0) {
  if(!std::isfinite(z) || !std::isfinite(a)) return;
  int iz = binIndexFromEdges(z, H.zedges);
  int ia = binIndexFromEdges(a, H.aedges);
  if (iz >= 0 && ia >= 0) {
    H.at(iz, ia) += weight;
    H.totalSamples += weight;
  }
}


// ---- Make an empty histogram with existing edges (no auto-ranging) ----
Hist2D makeEmptyHistWithEdges(const std::vector<float>& zEdges,
                              const std::vector<float>& aEdges) {
  if (zEdges.size() < 2 || aEdges.size() < 2) {
    throw std::runtime_error("makeEmptyHistWithEdges: edges too short");
  }
  Hist2D H;
  H.nz = int(zEdges.size()) - 1;
  H.na = int(aEdges.size()) - 1;
  H.zedges = zEdges;
  H.aedges = aEdges;
  H.counts.assign(size_t(H.nz) * size_t(H.na), 0.0f);
  H.totalSamples = 0.0f;
  return H;
}

// ---- raw-counts kth moment of 'a' per z-row (no density; uses bin centers) ----
static std::vector<float> kthMomentByZFromCounts(const Hist2D& H, int k) {
  std::vector<float> out(size_t(H.nz), std::numeric_limits<float>::quiet_NaN());
  for (int iz = 0; iz < H.nz; ++iz) {
    long double N = 0.0L, sumY = 0.0L;
    for (int ia = 0; ia < H.na; ++ia) {
      const long double n  = H.at(iz, ia);
      if (n <= 0.0L) continue;
      const long double ac = H.acenter(ia);
      const long double y  = std::pow(ac, k);
      sumY += n * y;
      N    += n;
    }
    if (N > 0.0L) out[size_t(iz)] = float(sumY / N);
  }
  return out;
}

// ---- Block-average: mean & SE over blocks, per z-bin (uses raw counts) ----
struct BlockMomentStats {
  std::vector<float> mean;  // size = nz
  std::vector<float> se;    // size = nz
};

BlockMomentStats blockAverageMomentByZ(
    const std::vector<std::vector<Angle>>& frames, // frames[t] = angles for that frame
    int k,
    int B,
    const std::vector<float>& zEdges,
    const std::vector<float>& aEdges)
{
  if (B < 2) throw std::runtime_error("blockAverageMomentByZ: B must be >= 2");
  const int F = int(frames.size());
  if (F < B) throw std::runtime_error("blockAverageMomentByZ: F < B");
  // Prepare per-block histograms with identical edges
  std::vector<Hist2D> HB;
  HB.reserve(B);
  for (int b = 0; b < B; ++b) HB.push_back(makeEmptyHistWithEdges(zEdges, aEdges));

  // Even split frames into B blocks: block b => [start[b], start[b+1])
  std::vector<int> start(B + 1, 0);
  for (int b = 0; b <= B; ++b) start[b] = (b * F) / B;

  // Accumulate raw counts per block
  for (int b = 0; b < B; ++b) {
    for (int f = start[b]; f < start[b + 1]; ++f) {
      hist2DAccumulate(HB[b], frames[size_t(f)], /*weight=*/1.0f);
    }
  }

  const int nz = int(zEdges.size()) - 1;
  BlockMomentStats out;
  out.mean.assign(size_t(nz), std::numeric_limits<float>::quiet_NaN());
  out.se  .assign(size_t(nz), std::numeric_limits<float>::quiet_NaN());

  std::vector<float> mb(size_t(B), std::numeric_limits<float>::quiet_NaN());
  std::vector<char>  has(size_t(B), 0);

  // For each z-row: compute block means, then mean & SE across blocks that have data
  for (int iz = 0; iz < nz; ++iz) {
    int M = 0;
    for (int b = 0; b < B; ++b) {
      // row count in this block
      long double Nrow = 0.0L;
      for (int ia = 0; ia < HB[b].na; ++ia) Nrow += HB[b].at(iz, ia);
      if (Nrow > 0.0L) {
        // kth moment for this row from this block
        long double N = 0.0L, sumY = 0.0L;
        for (int ia = 0; ia < HB[b].na; ++ia) {
          const long double n  = HB[b].at(iz, ia);
          if (n <= 0.0L) continue;
          const long double ac = HB[b].acenter(ia);
          const long double y  = std::pow(ac, k);
          sumY += n * y; N += n;
        }
        mb[size_t(b)]  = (N > 0.0L) ? float(sumY / N) : std::numeric_limits<float>::quiet_NaN();
        has[size_t(b)] = 1;
        ++M;
      } else {
        has[size_t(b)] = 0;
      }
    }

    if (M >= 1) {
      long double mu = 0.0L;
      for (int b = 0; b < B; ++b) if (has[size_t(b)]) mu += mb[size_t(b)];
      mu /= M;
      out.mean[size_t(iz)] = float(mu);

      if (M >= 2) {
        long double s2 = 0.0L;
        for (int b = 0; b < B; ++b) if (has[size_t(b)]) {
          const long double d = mb[size_t(b)] - mu;
          s2 += d * d;
        }
        out.se[size_t(iz)] = float(std::sqrt(s2 / (M * (M - 1))));
      } // else leave NaN
    }
  }
  return out;
}




// -------------- END STATISTICS ---------------

// -------------- START BASIC -------------------
float applyPBC(float x, float box){
  float hbox = box/2.0;
  float wrapped = fmod(x + hbox, box);
  if (wrapped < 0) wrapped += box;
  return wrapped - hbox;
}

float distance(const Atom& a, const Atom& b, const Box& box) {
  float dx, dy, dz, rsq;
  dx=applyPBC( a.x - b.x, box.Lx);
  dy=applyPBC( a.y - b.y, box.Ly);
  dz=applyPBC( a.z - b.z, box.Lz);
  rsq = dx*dx +dy*dy +dz*dz;
  return std::sqrt(rsq);
}

Angle computeTerminalAngle( Targets& target, const std::vector<Atom>& atoms, const Box& box) {
  Angle result;
  // compute C-C vector angle
  float dz;
  dz=applyPBC(atoms[target.t].z - atoms[target.c].z, box.Lz);
  result.ccCos = dz/distance(atoms[target.t], atoms[target.c], box);
  //std::cout << "target : " << target.t << " carbon : " << target.c << " dz " << dz << " result " << result.ccCos << "\n";

  // compute averaged C-H vector angle, assume CH bond length fixed
  float dx1, dx2, dx3, dy1, dy2, dy3, dz1, dz2, dz3, dx, dy;
  dx1 = applyPBC(atoms[target.h1].x - atoms[target.t].x, box.Lx);
  dx2 = applyPBC(atoms[target.h2].x - atoms[target.t].x, box.Lx);
  dx3 = applyPBC(atoms[target.h3].x - atoms[target.t].x, box.Lx);
  dx = dx1+dx2+dx3;

  dy1 = applyPBC(atoms[target.h1].y - atoms[target.t].y, box.Ly);
  dy2 = applyPBC(atoms[target.h2].y - atoms[target.t].y, box.Ly);
  dy3 = applyPBC(atoms[target.h3].y - atoms[target.t].y, box.Ly);
  dy = dy1 + dy2 + dy3;

  dz1 = applyPBC(atoms[target.h1].z - atoms[target.t].z, box.Lz);
  dz2 = applyPBC(atoms[target.h2].z - atoms[target.t].z, box.Lz);
  dz3 = applyPBC(atoms[target.h3].z - atoms[target.t].z, box.Lz);
  dz = dz1 + dz2 + dz3;

  float rsq = dx*dx + dy*dy + dz*dz;
  result.chCos = dz/std::sqrt(rsq);

  result.zloc = atoms[target.t].z;
  
  return result;
}




// ------------------------ MAIN ------------------------
int main(int argc, char** argv) {
  if (argc != 6 ) {
    std::cerr << "Error: Num Args not Match\n" ;
    std::cerr << "Usage : " << argv[0] << "dumpDir targetMol numMols timestep(ps) eqtime(ns)\n";
    return 1;
  }
  std::string dump = std::string(argv[1])  + argv[2] + ".dcd";
  int nummols = std::stoi(argv[3]);
  float timestep = std::stof(argv[4]);
  float eqtime = std::stof(argv[5]);

  // Read DCD file
  std::cout << "---- Reading dcd file at " << dump << " ----\n";
  DCDReader reader(dump);
  int numatoms = reader.natoms();
  std::cout << "number of atoms: " << numatoms << "\n";
  std::cout << "number of mols : " << nummols << "\n";
  std::cout << "number of frames (original): " << reader.nframes() << "\n";
  std::cout << "Initial " << eqtime << " ns are used for equilibration\n";
  int eqsnap=static_cast<int>(1000*eqtime/timestep);
  auto frames = reader.read_all();
  size_t numsnap = frames.size();
  std::cout << "number of frames (after EQ): " << numsnap - eqsnap << "\n\n";
  int oneMolecule = numatoms / nummols;


  // Read Lammps input file to get type, mass, mol etc
  std::string lammpsInput = std::string("../../../") + argv[2] +"/system.data";
  LammpsData data = readLammpsData(lammpsInput);

  std::cout << "---- Reading Lammps input at " << lammpsInput << " ----\n";
  std::cout << "Atoms style : " 
            << (data.atomsStyleHint.empty() ? "<none>" : data.atomsStyleHint) << "\n";
  std::cout << "Read atoms: " << data.atoms.size() << "\n";

  //generate type vector
  std::vector<int> atomTypes;
  if (!data.atoms.empty()) {
    for (size_t i = 0; i < data.atoms.size(); ++i) {
      const AtomRow& a = data.atoms[i];
      atomTypes.push_back(a.type);
    }
  }

  // Output video file
  std::ofstream ofs("video.lammpstrj");
  if (!ofs) {
    std::cerr << "Error. Cannot open the file\n";
    return 1;
  }

  std::cout << "Read masses: " << data.masses.size() << " types\n";
  std::vector<std::pair<int,double>> type2mass;
  if (!data.masses.empty()) {
    // print a map : type to mass 
    type2mass.reserve(data.masses.size());
    for (const auto& kv : data.masses) type2mass.emplace_back(kv.first, kv.second);
    std::sort(type2mass.begin(), type2mass.end(), [](const std::pair<int,double>& a, const std::pair<int,double>& b){ return a.first < b.first;});
    for (size_t i = 0; i < type2mass.size(); ++i) {
      std::cout << "  type " << type2mass[i].first << " mass " << type2mass[i].second << "\n";
    }
  }

  auto initL = findTargets(argv[2], true); // left
  auto initR = findTargets(argv[2], false); // right

  numsnap = numsnap-eqsnap;
  std::vector<Angle> angles; 
  angles.reserve(nummols*2*numsnap);
  for( size_t time=eqsnap; time<numsnap+eqsnap; time++ ) {
    auto frame = frames[time].atoms;
    const auto box = frames[time].box;
    // first, find pbc center of mass, shift it to the center of box
    float zc = comZPBC(frame, atomTypes, type2mass, box.Lz); 
    for ( auto& atom : frame ) atom.z = wrapPBC(atom.z - zc + box.Lz/2.0, box.Lz);
    // second, find geometric center of mass, shift box 
    float zc2 = comZ(frame, atomTypes, type2mass);
    for ( auto& atom : frame ) atom.z = wrapPBC(atom.z - zc2 + box.Lz/2.0, box.Lz);
    //std::cout << "Time " << time << " z center was " << zc << " now goes to " << zc2 << " \n";

    // find true center of mass, and shift again
    for( int mol=0; mol < nummols; mol++ ) {
      auto indexL = initL + static_cast<int>(oneMolecule*mol);
      auto indexR = initR + static_cast<int>(oneMolecule*mol);
      Angle left  = computeTerminalAngle(indexL, frame, box);
      Angle right = computeTerminalAngle(indexR, frame, box);
      angles.emplace_back(left);
      angles.emplace_back(right);
    }

    if (time % 100 == 0 ) {
      writeLammpsDump(ofs, time, frame, atomTypes,  box);
    }
  }
  ofs.close();

  /// makeHist2D(std::vector<Angle> data, int nz, int na)
  //auto H = makeHist2D(angles, 30, 50);

  /// makeHist2DSymZ(std::vector<Angle> data, float center, float Lz, float zBinWidth, int na)
  auto H =  makeHist2DSymZ(angles, 100, 200, 2, 50);
  hist2DAccumulate(H, angles);
  std::vector<float> raw = H.counts; 
  H.normalizeDensity();
  H.printHist("2dhist.dat");
  std::ofstream output("slice.dat");
  H.printZsliceHist(120, output);

  // generate terminal carbon density profile
  std::vector<float> rho(angles.size());
  std::transform(angles.begin(), angles.end(), rho.begin(), [](const Angle& a){ return static_cast<float>(a.zloc);});
  auto density = makeHistogramAuto(rho, 200);
  std::ofstream out("densityProfile.dat");
  printHistogram(density, out);

  auto stats1 = H.kthMomentStatsByZ(/*k=*/1, raw);
  auto stats3 = H.kthMomentStatsByZ(/*k=*/3, raw);
  std::ofstream output2("moments.dat");
  H.printMoments(output2, stats1, stats3);


  // try block averaging
  bool blockAverage=true;
  if(blockAverage) {
    std::vector<std::vector<Angle>> traj(numsnap, std::vector<Angle>(nummols*2));
    for( size_t time =0; time < numsnap; time++ ) {
      for ( int mol =0; mol< nummols*2; mol++ ) {
        traj[time][mol] = angles[time * (nummols*2) + mol];
      }
    }
    int B =20;
    auto stats1 = blockAverageMomentByZ(traj, 1, B, H.zedges, H.aedges);
    auto stats3 = blockAverageMomentByZ(traj, 3, B, H.zedges, H.aedges);
    std::ofstream output3("BAmoments.dat");
    H.printMoments(output3, *(momentStats*)&stats1, *(momentStats*)&stats3, 8);
  }






  return 0;
}

